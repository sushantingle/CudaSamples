#include "hip/hip_runtime.h"
﻿/**
This is test run to allocate zero copy memory.
Zero copy memory is memory mapped in device address space. This could be used to improve PCIe transfer.
We don't have to do explicit data transfer between host and device.

If device memory is frequently read/write, then it could be huge impact on performance,
since each operation needs to transferred to host on PCIe bus.
*/


#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <cstdlib>

__global__ void Zero_Copy_Add(float* a, float* b, float* c, int size)
{
	int gid = blockDim.x * blockIdx.x + threadIdx.x;

	if (gid < size)
	{
		c[gid] = a[gid] + b[gid];
	}
}

int main()
{
	int deviceId = 0;
	hipSetDevice(deviceId);

	// Get Device Properties
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, deviceId);
	if (!deviceProp.canMapHostMemory) // check if mapping is supported or not.
	{
		printf("Device %d does not support mapping CPU host memory.\n");
		return 0;
	}

	int size = 1 >> 25;
	int bytes = size * sizeof(float);

	dim3 block_size(128);
	dim3 grid(size / block_size.x);
	float* h_dataA, * h_dataB, * h_dataC;

	h_dataC = (float*)malloc(bytes);
	
	// allocate mapped host memory
	hipHostAlloc((float**)&h_dataA, bytes, hipHostMallocMapped);
	hipHostAlloc((float**)&h_dataB, bytes, hipHostMallocMapped);

	float* d_dataA, *d_dataB, *d_dataC;
	// Get Device pointer to mapped memory
	hipHostGetDevicePointer((float**)&d_dataA, h_dataA, 0);
	hipHostGetDevicePointer((float**)&d_dataB, h_dataB, 0);

	hipMalloc((float**)&d_dataC, bytes);

	Zero_Copy_Add << <grid, block_size >> > (d_dataA, d_dataB, d_dataC, size);

	hipDeviceSynchronize();
	
	hipMemcpy(h_dataC, d_dataC, bytes, hipMemcpyDeviceToHost);

	hipHostFree(h_dataA);
	hipHostFree(h_dataB);
	hipFree(d_dataC);
	free(h_dataC);

	hipDeviceReset();

	return 0;

}
