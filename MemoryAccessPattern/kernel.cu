#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>

// unaligned global load
__global__ void unaligned_memory_access(float* a, float* b, float* c, int size, int offset)
{
    int gid = blockDim.x * blockIdx.x + threadIdx.x;
    int inputIdx = gid + offset;
    if (inputIdx < size)
    {
        c[gid] = a[inputIdx] + b[inputIdx];
    }
}

// unaligned global store
__global__ void unaligned_memory_store(float* a, float* b, float* c, int size, int offset)
{
    int gid = blockDim.x * blockIdx.x + threadIdx.x;
    int outputIdx = gid + offset;
    if (outputIdx < size)
    {
        c[outputIdx] = a[gid] + b[gid];
    }
}

int main(int argc, char** argv)
{
    int size = 1 << 22;
    int block_size = 128;
    int bytes = size * sizeof(float);
    int offset = 0;
    int kernel = 0;
    if (argc > 1)
        offset = atoi(argv[1]);

    if (argc > 2)
        kernel = atoi(argv[2]);

    float* h_A, *h_B;
    h_A = (float*)malloc(bytes);
    h_B = (float*)malloc(bytes);

    float* d_A, * d_B, *d_C;
    hipMalloc((float**)&d_A, bytes);
    hipMalloc((float**)&d_B, bytes);
    hipMalloc((float**)&d_C, bytes);

    hipMemcpy(d_A, h_A, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, bytes, hipMemcpyHostToDevice);

    dim3 block(block_size);
    dim3 grid(size / block.x);

    switch (kernel)
    {
    case 1:
        unaligned_memory_access << <grid, block >> > (d_A, d_B, d_C, size, offset);
        break;
    case 2:
        unaligned_memory_store << <grid, block >> > (d_A, d_B, d_C, size, offset);
        break;
    }
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);

    hipDeviceReset();
    return 0;
}
