﻿
#include "hip/hip_runtime.h"


#include <stdio.h>

// When we allocate host memory and do HostToDevice Transfer, Cuda copies host data from Pagable memory to Pinned Memory.
// Since pagable memory can't be transferred directly to GPU, Pinned memory needs to be used.
// Below test program is to show how can we allocate pinned memory.

int main()
{
    int size = 1 << 22;
    int bytes = size * sizeof(float);

    float* h_data;
    float* d_data;

    // Allocate Pinned memory
    hipHostMalloc((float**) &h_data, bytes, hipHostMallocDefault);
    hipMalloc((float**)&d_data, bytes);

    hipMemcpy(d_data, h_data, bytes, hipMemcpyHostToDevice);
    hipMemcpy(h_data, d_data, bytes, hipMemcpyDeviceToHost);

    hipFree(d_data);
    hipHostFree(h_data);

    hipDeviceReset();
    return 0;
}
