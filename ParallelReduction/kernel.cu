#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <cstdlib>

__global__ void neighboured_pair_reduction(int* data, int* temp, int size)
{
    int tId = threadIdx.x;
    int gId = blockDim.x * blockIdx.x + threadIdx.x;

    if (gId > size)
        return;

    for (int offset = 1; offset <= blockDim.x / 2; offset *= 2)
    {
        if (tId % (2 * offset) == 0)
        {
            data[gId] += data[gId + offset];
        }
        __syncthreads();
    }

    if (tId == 0)
    {
        temp[blockIdx.x] = data[gId];
    }
}


int Reduction_CPU(int* data, int size)
{
    int result = 0;
    for (int i = 0; i < size; ++i)
    {
        result += data[i];
    }
    return result;
}

void Initialize(int* data, int size)
{
    for (int i = 0; i < size; ++i)
    {
        data[i] = i;
    }
}

int main()
{
    int size = 1 << 27; // 128MB size
    int byte_size = size * sizeof(int);
    int block_size = 128;

    int* h_data;
    int* h_ref;

    h_data = (int*)malloc(byte_size);
    Initialize(h_data, size);

    int CPU_Result = Reduction_CPU(h_data, size);

    dim3 block(block_size);
    dim3 grid(size / block.x);

    printf("kernel Launch Parameters : Grid %d      Block.x %d \n", grid.x, block.x);
    int partial_sum_array_size = sizeof(int) * grid.x;
    h_ref = (int*)malloc(partial_sum_array_size);

    int* d_input, *d_temp;
    hipMalloc((void**)&d_input, byte_size);
    hipMalloc((void**)&d_temp, partial_sum_array_size);

    hipMemset(d_temp, 0, partial_sum_array_size);
    hipMemcpy(d_input, h_data, byte_size, hipMemcpyHostToDevice);
    neighboured_pair_reduction << <grid, block >> > (d_input, d_temp, size);
    hipDeviceSynchronize();

    hipMemcpy(h_ref, d_temp, partial_sum_array_size, hipMemcpyDeviceToHost);

    int gpu_result = 0;
    for (int i = 0; i < grid.x; ++i)
    {
        gpu_result += h_ref[i];
    }

    if (CPU_Result == gpu_result)
    {
        printf("Results are same on CPU and GPU.\n");
    }

    hipFree(d_input);
    hipFree(d_temp);
    free(h_data);
    free(h_ref);

    hipDeviceReset();
    return 0;
}
